#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void hello_kernel()
{
	printf("Hello World from Thread %d\n", threadIdx.x);
}


int main(int argc, char *argv[])
{

	//set the CUDA device to the default CUDA GPU (device 0)
	hipError_t result = hipSetDevice(0);
	if (result != hipSuccess ){
		printf("Error setting default GPU device.\n");
	}
	
	//call a CUDA kernel
	dim3 blocksPerGrid(1, 1, 1);
	dim3 threadsPerBlock(10, 1, 1);
	hello_kernel<<<blocksPerGrid, threadsPerBlock>>>();
	
	//synchronise
	hipDeviceSynchronize();
	
	return 0;
}